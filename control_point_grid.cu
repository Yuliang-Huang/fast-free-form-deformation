#include "control_point_grid.h"

// Constructor
ControlPointGrid::ControlPointGrid(int w, int h, int d, int nchannels) 
{
    this->width = w;
    this->height = h;
    this->depth = d;
    this->nchannels = nchannels;

    hipMallocManaged(&this->texArray, nchannels * sizeof(hipTextureObject_t));//https://forums.developer.nvidia.com/t/cuda-passing-a-class-to-a-kernel/219779/2
    this->dataArray = new hipArray*[nchannels];

    for (int i = 0; i<nchannels; i++)
    {
        this->create3DTexture(this->dataArray[i], this->texArray[i]);
    }
}

// Destructor
ControlPointGrid::~ControlPointGrid() 
{
    for (int i = 0; i<this->nchannels; i++)
    {
        hipDestroyTextureObject(this->texArray[i]);
        hipFreeArray(this->dataArray[i]);            
    }
}

// Uploads device data to 3D texture
void ControlPointGrid::copyData(torch::Tensor &data) {

    if (data.size(0)!=this->nchannels || data.size(1)!=this->depth || data.size(2)!=this->height || data.size(3)!=this->width)
    {
        std::cerr << "ERROR input tensor shape mismatched with Texture Container extent!" << std::endl;
    }

    hipMemcpy3DParms copyParams = {0};

    // Upload X displacement from device memory
    copyParams.extent = make_hipExtent(this->width, this->height, this->depth);
    copyParams.kind = hipMemcpyDeviceToDevice;
    for (int i = 0; i<this->nchannels; i++)
    {
        copyParams.srcPtr = make_hipPitchedPtr(data.index({i}).data_ptr<float>(), this->width * sizeof(float), this->width, this->height);
        copyParams.dstArray = this->dataArray[i];
        hipMemcpy3D(&copyParams);
    }
}

// Creates a 3D texture and surface object
void ControlPointGrid::create3DTexture(hipArray_t& d_array, hipTextureObject_t& tex) {
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipExtent volumeSize = make_hipExtent(this->width, this->height, this->depth);
    
    // Allocate 3D CUDA array
    hipMalloc3DArray(&d_array, &channelDesc, volumeSize);

    // Create texture object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = d_array;

    hipTextureDesc texDesc = {};
    texDesc.normalizedCoords = false;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.addressMode[2] = hipAddressModeBorder;
    texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr);
}
